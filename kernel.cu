﻿#include "Tensor.h"
#include <iostream>
#include <stdio.h>
#include <string>
using namespace std;

int main()
{ // ======== Test Input Setup for Broadcasted Elementwise MatMul ========
    int shape_input[] = {2, 6};
    int data_input[12] = {1, 3, 5, 7, 9, 11, 2, 4, 6, 8, 10, 12};
    Tensor<int> array1(data_input, shape_input, 2);

    int shape_weights[] = {3, 6};
    int data_weights[18] = {1, 0, 1, 0, 1, 0, 0, 1, 0, 1, 0, 1, 2, 2, 2, 2, 2, 2};
    Tensor<int> array2(data_weights, shape_weights, 2);

    // Perform broadcasted matmul
    Tensor<int> array3 = Tensor<int>::matMul(array1, array2);

    // Print output
    std::cout << array3.print();
    return 0;
}
